#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/core/operator.h"
#include "spatial_sigmoid_op.h"

namespace caffe2 {

namespace {


__global__ void SpatialSigmoidKernel(const int N,
    const float* logits, const float* targets,float* loss) {
  CUDA_1D_KERNEL_LOOP(index, N) {
      loss[index]=-1. * logits[index] * (targets[index] - (logits[index] >= 0)) +
          logf(1 + expf(logits[index] - 2 * logits[index] * (logits[index] >= 0)));
  }
}
}  // namespace

template <>
bool SpatialSigmoidOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& Y = Input(1);
  
  int N = X.size();
  auto* loss = Output(0);

  loss->ResizeLike(X);


  SpatialSigmoidKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N,X.data<float>(), Y.data<float>(), loss->mutable_data<float>());

  return true;
}

REGISTER_HIP_OPERATOR(SpatialSigmoid,
                       SpatialSigmoidOp<float, HIPContext>);

}